#include "hip/hip_runtime.h"

#include <caffepro/layers/softmax_loss_layer.h>
#include <caffepro/layers/softmax_layer.h>
#include <caffepro/proto/caffe.pb.h>
#include <caffepro/math/cublas_wrapper.h>
#include <caffepro/utils/utils.h>
#include <caffepro/context/common_names.h>
#include <caffepro/utils/string_uitls.h>

#include <functional>

#define MAX_MINIBATCH 512

namespace caffepro {
	softmax_loss_layer::softmax_loss_layer(caffepro_context *context, const LayerParameter &param)
		: caffepro_layer(context, param) {

		attr_.num_inputs_min = attr_.num_inputs_max = 2;
		attr_.num_outputs_min = attr_.num_outputs_max = 1;

		attr_.set_constraint(
			layer_attribute::CF_REQUIRE_UNIQUE_DEVICE
			| layer_attribute::CF_REQUIRE_SAME_NUM
			| layer_attribute::CF_REQUIRE_SAME_DEVICE
			| layer_attribute::CF_REQUIRE_NDIM_4
			| layer_attribute::CF_REQUIRE_FIXEDLEN_DIM // remove it in the future
			);

		attr_.usage = layer_attribute::USAGE_LOSS;
	}

	softmax_loss_layer::~softmax_loss_layer() {
		release_all();
	}

	void softmax_loss_layer::init() {
		check_input();
		output_top_n_ = 0;

		coeff_ = (data_type)1.f;
		if (layer_param_.has_loss_param() && layer_param_.loss_param().has_coeff()) {
			coeff_ = layer_param_.loss_param().coeff();
		}

		softmax_inputs_.resize(1);
		softmax_inputs_[0] = inputs_[0];
		prob_.reset(new node_blob());
		softmax_outputs_.resize(1);
		softmax_outputs_[0] = prob_;
		
		softmax_.reset(new softmax_layer(context_, layer_param_));
		softmax_->bind(softmax_inputs_, softmax_outputs_);
		softmax_->init();

		correct_.reset(new node_blob());
		sum_multiplier_.reset(new node_blob());
		avg_prob_.reset(new node_blob());
	}

	void softmax_loss_layer::resize() {
		check_input();

		softmax_->resize();
		CHECK(inputs_[0]->get(0)->same_dim_at(2));
		CHECK(inputs_[1]->get(0)->count() == inputs_[0]->get(0)->num());

		if (outputs_[0]->size() == 0) {

			int num_of_outputs = 2;
			if (!context_->get_global_cfg(GLOBALCFGNAME_SHOW_TOP_N_ACC).empty()) {
				num_of_outputs++;
				output_top_n_ = string_to_int(context_->get_global_cfg(GLOBALCFGNAME_SHOW_TOP_N_ACC));
				CHECK_GT(output_top_n_, 0);
			}

			outputs_[0]->add(boost::shared_ptr<device_blob>(
				device_blob::create_4d(context_, 1, num_of_outputs, 1, 1, inputs_[0]->get(0)->device_id())
				));

			outputs_[0]->tags().resize(num_of_outputs);
			outputs_[0]->tags()[0] = "Error";
			outputs_[0]->tags()[1] = "Loss";
			if (num_of_outputs >= 3) {
				outputs_[0]->tags()[2] = "Top " + std::to_string(output_top_n_) + " error";
			}

			avg_prob_->add(boost::shared_ptr<device_blob>(
				device_blob::create_4d(
					context_,
					inputs_[0]->get(0)->num(),
					inputs_[0]->get(0)->dim_at(2),
					1,
					1,
					inputs_[0]->get(0)->device_id()
					)
				));

			correct_->add(boost::shared_ptr<device_blob>(
				device_blob::create_4d(
					context_, inputs_[0]->get(0)->num(), 1, 1, 1, inputs_[0]->get(0)->device_id()
					)
				));

			sum_multiplier_->add(boost::shared_ptr<device_blob>(
				device_blob::create_4d(
					context_,
					1,
					1,
					inputs_[0]->get(0)->height(),
					inputs_[0]->get(0)->width(),
					inputs_[0]->get(0)->device_id()
					)
				));

			sum_multiplier_->get(0)->fill_data((data_type)1.f);
		}
		else if (inputs_[0]->reshaped()) {
			// do not need to reshape top

			avg_prob_->get(0)->reshape_4d(
				inputs_[0]->get(0)->num(),
				inputs_[0]->get(0)->channels(),
				1,
				1
				);

			correct_->get(0)->reshape_4d(inputs_[0]->get(0)->num(), 1, 1, 1);
			
			sum_multiplier_->get(0)->reshape_4d(1, 1, inputs_[0]->get(0)->height(), inputs_[0]->get(0)->width());
			sum_multiplier_->get(0)->fill_data((data_type)1.f);
		}
	}

	__global__ static void calc_acc_gpu_kernel(const int n, const int dim, const data_type *probs,
		const data_type *label, data_type *correct, data_type *acc, data_type *loss) {
		__shared__ int ans[MAX_MINIBATCH];
		__shared__ data_type ans_loss[MAX_MINIBATCH];

		int cur_ins = threadIdx.x;
		probs += cur_ins * dim;

		data_type max_v = -1;
		int max_index = -1;
		for (int i = 0; i < dim; i++) {
			if (probs[i] > max_v) {
				max_v = probs[i];
				max_index = i;
			}
		}

		int ground_truth = (label[cur_ins] >= 0 ? (int)(label[cur_ins] + 0.5) : -1);
		correct[cur_ins] = max_index;
		ans[cur_ins] = (ground_truth == max_index) ? 1 : 0;
		ans_loss[cur_ins] = (ground_truth >= 0 ? -__logf(probs[ground_truth]) : 0);

		__syncthreads();

		if (cur_ins == 0) {
			data_type n_correct = 0;
			data_type ls = 0;
			for (int i = 0; i < n; i++) {
				n_correct += ans[i];
				ls += ans_loss[i];
			}
			acc[0] = 1 - n_correct / n;
			loss[0] = ls / n;
		}
	}

	__global__ void static bp_softlost_gpu_kernel(const int n, const int feature_dim, const int spatial_size,
		const data_type *probs, const data_type *label,
		data_type *diff, const data_type coeff, const data_type scale_targets) {

		CUDA_KERNEL_LOOP(index, n) {
			int dest_idx = index;

			index /= spatial_size;
			int cur_ins = index / feature_dim;
			int cur_dim = index % feature_dim;
			int total_ins = n / feature_dim / spatial_size;

			int ground_truth = (label[cur_ins] >= 0 ? (int)(label[cur_ins] + 0.5) : -1);

			if (ground_truth >= 0) {
				if (ground_truth == cur_dim) {
					data_type v = coeff * (probs[dest_idx] - 1) / (data_type)total_ins / (data_type)spatial_size;
					if (scale_targets == 0) {
						diff[dest_idx] = v;
					}
					else {
						diff[dest_idx] = diff[dest_idx] * scale_targets + v;
					}
				}
				else {
					data_type v = coeff * probs[dest_idx] / (data_type)total_ins / (data_type)spatial_size;
					if (scale_targets == 0) {
						diff[dest_idx] = v;
					}
					else {
						diff[dest_idx] = diff[dest_idx] * scale_targets + v;
					}
				}
			}
			else {
				if (scale_targets == 0) {
					diff[dest_idx] = 0;
				}
				// do not need to modify the diff when scale_target != 0
			}
		}
	}

	void softmax_loss_layer::on_forward(int device_index) {
		// The forward pass computes the softmax prob values.
		softmax_->forward();

		// average up
		const int spatial_size = inputs_[0]->get(device_index)->width() * inputs_[0]->get(device_index)->height();
		const int num = inputs_[0]->get(device_index)->num();
		const int feature_dim = inputs_[0]->get(device_index)->dim_at(2);

		CHECK_GT(spatial_size, 0);

		if (spatial_size > 1) {
			cublas_wrapper<data_type> cublas(context_, context_->get_current_device()->device_id());
			cublas.gemv(
				CblasNoTrans,
				num * feature_dim,
				spatial_size,
				(data_type)1. / (data_type)spatial_size,
				prob_->get(device_index)->gpu_data(),
				sum_multiplier_->get(device_index)->gpu_data(),
				(data_type)0.,
				avg_prob_->get(device_index)->mutable_gpu_data()
				);
		}
		else {
			avg_prob_->get(device_index)->copy_data_from_via_gpu(*prob_->get(device_index));
		}

		CHECK_LE(num, MAX_MINIBATCH);

		//auto ptr = avg_prob_->get(device_index)->cpu_data();
		//auto ptr2 = inputs_[1]->get(device_index)->cpu_data();

		calc_acc_gpu_kernel<<<1, num>>>(
			num, 
			feature_dim, 
			avg_prob_->get(device_index)->gpu_data(),
			inputs_[1]->get(device_index)->gpu_data(), 
			correct_->get(device_index)->mutable_gpu_data(),
			outputs_[0]->get(device_index)->mutable_gpu_data(), 
			outputs_[0]->get(device_index)->mutable_gpu_data() + 1
			);

		// optional: calc top_n error
		// cpu code
		if (output_top_n_ >= 1) {
			const data_type *avg_prob_data = avg_prob_->get(device_index)->cpu_data();
			const data_type *label_data = inputs_[1]->get(device_index)->cpu_data();
			std::vector<std::pair<data_type, int> > score(feature_dim);

			int top_n_acc = 0;
			for (int i = 0; i < num; i++, avg_prob_data += feature_dim) {
				for (int k = 0; k < feature_dim; k++) {
					score[k] = std::make_pair(avg_prob_data[k], k);
				}
				std::sort(score.begin(), score.end(), std::greater<std::pair<data_type, int> >());

				int label_id = (int)(label_data[i] + 0.5f);
				for (int k = 0; k < output_top_n_ && k < feature_dim; k++) {
					if (score[k].second == label_id) {
						top_n_acc++;
						break;
					}
				}
			}

			outputs_[0]->get(device_index)->mutable_cpu_data()[2] = 1.f - (data_type)top_n_acc / num;
		}
	}

	void softmax_loss_layer::on_backward(int device_index, act_selector bp_acts, weight_selector bp_weights, act_selector clear_acts_diff, weight_selector clear_weights_diff) {
		const data_type beta_acts = get_beta(clear_acts_diff, 0);
		hipdnnHandle_t handle = context_->get_current_device()->cudnn_handle();

		if (should_bp(bp_acts, 0)) {
			const int count = prob_->get(device_index)->count();
			const int feature_dim = prob_->get(device_index)->channels();
			const int spatial_size = prob_->get(device_index)->width() * prob_->get(device_index)->height();

			KERNEL_CALL(bp_softlost_gpu_kernel, count)(
				count,
				feature_dim,
				spatial_size,
				prob_->get(device_index)->gpu_data(),
				inputs_[1]->get(device_index)->gpu_data(),
				inputs_[0]->get(device_index)->mutable_gpu_diff(),
				coeff_,
				beta_acts
				);
		}
	}
}