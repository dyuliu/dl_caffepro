#include "hip/hip_runtime.h"

#include <caffepro/layers/softthr_layer.h>
#include <caffepro/utils/utils.h>
#include <caffepro/proto/caffe.pb.h>

namespace caffepro {
	softthr_layer::softthr_layer(caffepro_context *context, const LayerParameter &param)
		: caffepro_layer(context, param) {

		attr_.num_inputs_min = attr_.num_inputs_max = 1;
		attr_.num_outputs_min = attr_.num_outputs_max = 1;

		attr_.set_constraint(
			layer_attribute::CF_REQUIRE_SAME_DIMTYPE_ACROSS_DEVICES
			| layer_attribute::CF_ALLOW_INPLACE
			| layer_attribute::CF_FORBID_INPLACE_USAGE_NEXT_WHEN_INPLACE
			);
	}

	softthr_layer::~softthr_layer() {
		release_all();
	}

	__global__ void softthr_forward(const int n, const data_type* in, data_type* out, float softthr_thr) {
		CUDA_KERNEL_LOOP(index, n) {
			//out[index] = in[index] > 0 ? in[index] : in[index] * softthr_thr;
			out[index] = in[index] > softthr_thr ? (in[index] - softthr_thr) : (in[index] < -softthr_thr ? (in[index] + softthr_thr) : 0);
		}
	}

	void softthr_layer::on_forward(int device_index) {
		const data_type* bottom_data = inputs_[0]->get(device_index)->gpu_data();
		data_type* top_data = outputs_[0]->get(device_index)->mutable_gpu_data();
		const int count = inputs_[0]->get(device_index)->count();

		KERNEL_CALL(softthr_forward, count)(count, bottom_data, top_data, layer_param_.softthr_param().softthr_thr());
		CUDA_POST_KERNEL_CHECK;
	}

	__global__ void softthr_backward(const int n, const data_type* in_diff,
		const data_type* in_data, data_type* out_diff, float softthr_thr, const data_type scale_target) {
		CUDA_KERNEL_LOOP(index, n) {
			//data_type v = in_diff[index] * (in_data[index] > 0 ? 1 : softthr_thr);
			data_type v = in_diff[index] * (in_data[index] > softthr_thr ? 1 : (in_data[index] < -softthr_thr ? 1 : 0));
			if (scale_target == 0) {
				out_diff[index] = v;
			}
			else {
				out_diff[index] = out_diff[index] * scale_target + v;
			}
		}
	}

	void softthr_layer::on_backward(int device_index, act_selector bp_acts, weight_selector bp_weights, act_selector clear_acts_diff, weight_selector clear_weights_diff) {
		const data_type beta_acts = get_beta(clear_acts_diff, 0);

		if (should_bp(bp_acts, 0)) {
			const data_type* bottom_data = inputs_[0]->get(device_index)->gpu_data();
			const data_type* top_diff = outputs_[0]->get(device_index)->gpu_diff();
			data_type* bottom_diff = inputs_[0]->get(device_index)->mutable_gpu_diff();
			const int count = inputs_[0]->get(device_index)->count();

			KERNEL_CALL(softthr_backward, count)(count, top_diff, bottom_data, bottom_diff, layer_param_.softthr_param().softthr_thr(), beta_acts);
			CUDA_POST_KERNEL_CHECK;
		}
	}
}